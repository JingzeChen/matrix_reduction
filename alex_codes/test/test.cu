#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include "iostream"

__global__ void test_min(int * min_num) {
    int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    if (thread_id < 5) {
        return;
    }
    printf("%d %d\n", *min_num, thread_id);
    atomicMin(min_num, thread_id);
}

int main(int argc, char * argv[]) {
    int * min_num;
    int num = 220;
    hipMalloc((void **) &min_num, sizeof(int));
    hipMemcpy(min_num, &num, sizeof(int), hipMemcpyHostToDevice);
    test_min <<<1000, 1>>> (min_num);
    hipMemcpy(&num, min_num, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << num << std::endl;;
}
