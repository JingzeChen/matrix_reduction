#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <>
#include "gpu_boundary_matrix.h"
#define BLOCK_BITS 4

typedef long indx;
typedef short dimension;

__global__ void test_length(size_t * column_length, int column_num) {
    int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    if (thread_id >= column_num)
        return;
}

__global__ void allocate_all_columns(indx ** tmp_gpu_columns, size_t * column_length, int column_num,
        ScatterAllocator::AllocatorHandle allocator) {
    int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    if (thread_id >= column_num)
        return;

    auto length = column_length[thread_id];
    tmp_gpu_columns[thread_id] = (indx *) allocator.malloc(sizeof(indx) * length);
}

__global__ void transform_all_columns(indx ** tmp_gpu_columns, size_t * column_length, column *matrix, int column_num,
        ScatterAllocator::AllocatorHandle allocator) {
    int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    if (thread_id >= column_num)
        return;

    auto src_length = column_length[thread_id];
    auto src_data = tmp_gpu_columns[thread_id];
    auto col = &matrix[thread_id];
    col->data_length = 0;
    indx last_pos = -1;
    for (size_t i = 0; i < src_length; i++) {
        indx current_pos = src_data[i] / BLOCK_BITS;
        if (last_pos != current_pos) {
            col->data_length++;
            last_pos = current_pos;
        }
    }

    col->pos = (indx *) allocator.malloc(sizeof(indx) * col->data_length);
    col->value = (unsigned long long *) allocator.malloc(sizeof(unsigned long long) * col->data_length);

    last_pos = -1;
    unsigned long long last_value = 0;
    int cur_block_id = 0;

    for (int i = 0; i < src_length; i++) 
    {
        indx current_pos = src_data[i] / BLOCK_BITS;
        if (last_pos != current_pos) {
            if (last_pos != -1) {
                col->pos[cur_block_id] = last_pos;
                col->value[cur_block_id] = last_value;
                cur_block_id++;
            }
	    last_pos = current_pos;
            last_value = 0;
        }
        unsigned long long mask = ((unsigned long long) 1) << (src_data[i] % BLOCK_BITS);
        last_value |= mask;
	    if(i == (src_length-1))
	    {
		    col->pos[cur_block_id] = last_pos;
		    col->value[cur_block_id] = last_value;
	    }
    }
}

gpu_boundary_matrix::gpu_boundary_matrix(phat::boundary_matrix <phat::vector_vector> *src_matrix,
                                         indx chunks_num, ScatterAllocator::AllocatorHandle allocator) {
    auto cols_num = (size_t) src_matrix->get_num_cols();
    auto h_matrix = new column[cols_num];
    auto h_chunk_offset = new indx[cols_num];
    auto h_column_type = new short[cols_num];
    auto h_column_length = new size_t[cols_num];
    auto h_dims = new dimension[cols_num];
    auto h_lowest_one_lookup = new indx[cols_num];
    auto h_chunks_start_offset = new indx[chunks_num + 1];

    auto chunk_size = (size_t) CUDA_THREADS_EACH_BLOCK(cols_num);

    for (phat::index i = 0, chunk_pos = 0; i < cols_num; i++) {
        phat::column col;
        src_matrix->get_col(i, col);
        h_column_length[i] = col.size();
        h_column_type[i] = GLOBAL;
        h_lowest_one_lookup[i] = -1;
        h_dims[i] = src_matrix->get_dim(i);
        if (i % chunk_size == 0) {
            h_chunks_start_offset[chunk_pos] = i;
            chunk_pos++;
        }
    }

    h_chunk_offset[chunks_num] = (indx)(cols_num);

    gpuErrchk(hipMalloc((void **) &matrix, sizeof(column) * cols_num));
    gpuErrchk(hipMalloc((void **) &chunk_offset, sizeof(indx) * (chunks_num + 1)));
    gpuErrchk(hipMalloc((void **) &chunk_columns_finished, sizeof(indx) * chunks_num));
    gpuErrchk(hipMalloc((void **) &column_type, sizeof(short) * cols_num));
    gpuErrchk(hipMalloc((void **) &dims, sizeof(dimension) * cols_num));
    gpuErrchk(hipMalloc((void **) &column_length, sizeof(size_t) * cols_num));
    gpuErrchk(hipMalloc((void **) &lowest_one_lookup, sizeof(indx) * cols_num));
    gpuErrchk(hipMalloc((void **) &is_active, sizeof(bool) * cols_num));
    gpuErrchk(hipMalloc((void **) &is_ready_for_mark, sizeof(bool) * cols_num));

    size_t * d_column_length;
    gpuErrchk(hipMalloc((void **) &d_column_length, sizeof(size_t) * cols_num));
    gpuErrchk(hipMemcpy(d_column_length, h_column_length, sizeof(size_t) * cols_num, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(dims, h_dims, sizeof(dimension) * cols_num, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(column_length, h_column_length, sizeof(size_t) * cols_num, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(lowest_one_lookup, h_lowest_one_lookup, sizeof(indx) * cols_num, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(chunk_offset, h_chunk_offset, sizeof(indx) * (chunks_num + 1), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(column_type, h_column_type, sizeof(short) * cols_num, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(matrix, h_matrix, sizeof(column) * cols_num, hipMemcpyHostToDevice));

    indx ** tmp_gpu_columns, ** h_tmp_gpu_columns;
    h_tmp_gpu_columns = new indx * [cols_num];
    gpuErrchk(hipMalloc((void **) &tmp_gpu_columns, sizeof(indx *) * cols_num));
    allocate_all_columns <<< CUDA_BLOCKS_NUM(cols_num), CUDA_THREADS_EACH_BLOCK(cols_num) >>> (tmp_gpu_columns,
            d_column_length, cols_num, allocator);
    hipMemcpy(h_tmp_gpu_columns, tmp_gpu_columns, sizeof(indx *) * cols_num, hipMemcpyDeviceToHost);

    for (phat::index i = 0; i < cols_num; i++) {
        phat::column col;
        src_matrix->get_col(i, col);
        auto col_data_ptr = &col[0];

        gpuErrchk(hipMemcpy(h_tmp_gpu_columns[i], col_data_ptr, sizeof(indx) * col.size(),
                             hipMemcpyHostToDevice));
    }

    transform_all_columns <<< CUDA_BLOCKS_NUM(cols_num), CUDA_THREADS_EACH_BLOCK(cols_num) >>> (tmp_gpu_columns,
            column_length, matrix, cols_num, allocator);

    gpuErrchk(hipFree(tmp_gpu_columns));
    delete[] h_matrix;
    delete[] h_chunk_offset;
    delete[] h_column_type;
    delete[] h_column_length;
    delete[] h_dims;
    delete[] h_lowest_one_lookup;
    delete[] h_chunks_start_offset;
    delete[] h_tmp_gpu_columns;
}

/*__host__ __device__
gpu_boundary_matrix::~gpu_boundary_matrix(phat::boundary_matrix<phat::vector_vector> *src_matrix,
                                          column *d_matrix, , std::vector<indx> &lowest_one_lookup, std::vector<short> &column_type)
{
    indx cols_num = src_matrix->get_num_cols();
    auto h_all_columns = new column[cols_num];

    gpuErrchk(hipMemcpy(h_matrix, d_matrix, sizeof(column), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_all_columns, h_matrix->data, sizeof(column) * cols_num, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&lowest_one_lookup[0], h_matrix->lowest_one_lookup, sizeof(indx) * cols_num,
                     hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&column_type[0], h_matrix->column_type, sizeof(short) * cols_num, hipMemcpyDeviceToHost));

    for (int i = 0; i < cols_num; i++) {
        auto h_single_column = h_all_columns[i];
        if (h_single_column.data_length == 0 || column_type[i] != GLOBAL) {
            continue;
        }

    phat::column tmp_vector(h_single_column.data_length);
    gpuErrchk(hipMemcpy(&tmp_vector[0], h_single_column.data, sizeof(indx) * h_single_column.data_length, hipMemcpyDeviceToHost));
    src_matrix->set_col(i, tmp_vector);
    }
}*/

__device__ dimension get_dim(dimension* dims, int col_id) {
    return dims[col_id];
}

__device__ bool is_empty(column* matrix, int col_id) {
    return matrix[col_id].data_length == 0;
}

__device__ indx get_max_index(column* matrix, int col_id) {
    if (matrix[col_id].data_length == 0)
        return -1;
    else {
        unsigned long long t = matrix[col_id].value[matrix[col_id].data_length - 1];
        int cnt = 0;
        while (t >> 1 != 0) {
            t = t >> 1;
            cnt++;
        }
        return (matrix[col_id].pos[matrix[col_id].data_length - 1] * BLOCK_BITS + cnt);
    }
}

__device__ void clear_column(column* matrix, int col_id) {
    matrix[col_id].data_length = 0;
}

__device__ void remove_max_index(column* matrix, int col) {
    if(matrix[col].data_length == 0)
        return;
    unsigned long long t = matrix[col].value[matrix[col].data_length - 1];
    int cnt = 1;
    while (t >> 1 != 0) {
        t = t >> 1;
        cnt++;
    }
    int tx = (1 << (cnt-1));
    matrix[col].value[matrix[col].data_length - 1]  ^= tx;
    if (matrix[col].value[matrix[col].data_length - 1] == 0)
        matrix[col].data_length--;
}

__device__ void check_lowest_one_locally(column* matrix, short* column_type, indx* chunk_columns_finished, dimension* dims,indx my_col_id, indx block_id, indx chunk_start, indx row_begin, dimension cur_dim, indx *target_col, bool *ive_added) {
    if (cur_dim != get_dim(dims, my_col_id) || column_type[my_col_id] != GLOBAL) {
        if (!*ive_added) {
            atomicAdd((unsigned long long *) &chunk_columns_finished[block_id], (unsigned long long) 1);
            *ive_added = true;
        }
        return;
    }

    indx my_lowest_one = get_max_index(matrix, my_col_id);
    if (my_lowest_one >= row_begin) {
        for (indx col_id = chunk_start; col_id < my_col_id; col_id++) {
            indx this_lowest_one = get_max_index(matrix, col_id);
            if (this_lowest_one == my_lowest_one) {
                *target_col = col_id;
                if (*ive_added) {
                    atomicAdd((unsigned long long *) &chunk_columns_finished[block_id], (unsigned long long) -1);
                }
                return;
            }
        }
        if (!*ive_added) {
            atomicAdd((unsigned long long *) &chunk_columns_finished[block_id], (unsigned long long) 1);
            *ive_added = true;
        }
    }
}

__device__ void add_two_columns(column* matrix, int target, int source, ScatterAllocator::AllocatorHandle allocator) {
    size_t tgt_id = 0; size_t src_id = 0; size_t temp_id = 0;
    int msize = round_up_to_2s(matrix[target].data_length + matrix[source].data_length);
    auto new_pos = (indx *) allocator.malloc(sizeof(indx) * msize);
    auto new_value = (unsigned long long *) allocator.malloc(sizeof(unsigned long long) * msize);
    while (tgt_id < matrix[target].data_length && src_id < matrix[source].data_length) {
        if (matrix[target].pos[tgt_id] == matrix[source].pos[src_id]) {
            if (matrix[target].value[tgt_id] ^ matrix[source].value[src_id] != 0) {
                new_pos[temp_id] = matrix[target].pos[tgt_id];
                new_value[temp_id] = matrix[target].value[tgt_id] ^ matrix[source].value[src_id];
            }
            tgt_id++;
            src_id++;
            temp_id++;
        } else if (matrix[target].pos[tgt_id] < matrix[source].pos[src_id]) {
            if (matrix[target].pos[tgt_id] == matrix[source].pos[src_id + 1])
                tgt_id++;
            else {
                new_value[temp_id] = matrix[target].value[tgt_id];
                new_pos[temp_id] = matrix[target].pos[tgt_id];
                tgt_id++;
                temp_id++;
            }
        } else {
            if (matrix[target].pos[tgt_id + 1] == matrix[source].pos[src_id])
                src_id++;
            else {
                new_value[temp_id] = matrix[source].value[src_id];
                new_pos[temp_id] = matrix[source].pos[src_id];
                src_id++;
                temp_id++;
            }
        }
    }

    if (src_id < matrix[source].data_length) {
        memcpy(&new_value[temp_id], &matrix[source].value[src_id],
               sizeof(indx) * (matrix[source].data_length - src_id));
        memcpy(&new_pos[temp_id], &matrix[source].pos[temp_id],
               sizeof(indx) * (matrix[source].data_length - src_id));
        temp_id += matrix[source].data_length - src_id;
    } else if (tgt_id < matrix[target].data_length) {
        memcpy(&new_value[temp_id], &matrix[target].value[tgt_id],
               sizeof(indx) * (matrix[target].data_length - tgt_id));
        memcpy(&new_pos[temp_id], &matrix[target].pos[tgt_id],
               sizeof(indx) * (matrix[target].data_length - tgt_id));
        temp_id += matrix[target].data_length - tgt_id;
    }
    matrix[target].pos = new_pos;
    matrix[target].value = new_value;
    matrix[target].data_length = temp_id;
}

__device__ void mark_and_clean(column* matrix, indx* lowest_one_lookup, short* column_type, dimension* dims, indx my_col_id, indx row_begin, dimension cur_dim) {
    if (cur_dim != get_dim(dims, my_col_id) || column_type[my_col_id] != GLOBAL) {
        return;
    }

    indx my_lowest_one = get_max_index(matrix, my_col_id);
    if (lowest_one_lookup[my_lowest_one] == -1 && my_lowest_one >= row_begin) {
      	lowest_one_lookup[my_lowest_one] = my_col_id;
        column_type[my_col_id] = LOCAL_NEGATIVE;
        column_type[my_lowest_one] = LOCAL_POSITIVE;
        clear_column(matrix, my_lowest_one);
    }
}

