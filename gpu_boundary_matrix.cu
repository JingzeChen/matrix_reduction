#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <>
#include "gpu_boundary_matrix.h"
#define BLOCK_BITS 4

typedef long indx;
typedef short dimension;

__global__ void test_length(size_t * column_length, int column_num) {
    int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    if (thread_id >= column_num)
        return;
}

__global__ void allocate_all_columns(indx ** tmp_gpu_columns, size_t * column_length, int column_num,
        ScatterAllocator::AllocatorHandle allocator) {
    int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    if (thread_id >= column_num)
        return;

    auto length = column_length[thread_id];
    tmp_gpu_columns[thread_id] = (indx *) allocator.malloc(sizeof(indx) * length);
}

__global__ void transform_all_columns(indx ** tmp_gpu_columns, size_t * column_length, column *matrix, int column_num, bool* is_reduced, indx* lowest_one_lookup,
        ScatterAllocator::AllocatorHandle allocator) {
    int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    if (thread_id >= column_num)
        return;

    auto src_length = column_length[thread_id];
    auto src_data = tmp_gpu_columns[thread_id];
    auto col = &matrix[thread_id];
    col->data_length = 0;
    indx last_pos = -1;
    for (size_t i = 0; i < src_length; i++) {
        indx current_pos = src_data[i] / BLOCK_BITS;
        if (last_pos != current_pos) {
            col->data_length++;
            last_pos = current_pos;
        }
    }

    col->pos = (indx *) allocator.malloc(sizeof(indx) * col->data_length);
    col->value = (unsigned long long *) allocator.malloc(sizeof(unsigned long long) * col->data_length);

    last_pos = -1;
    unsigned long long last_value = 0;
    int cur_block_id = 0;

    for (int i = 0; i < src_length; i++)
    {
        indx current_pos = src_data[i] / BLOCK_BITS;
        if (last_pos != current_pos) {
            if (last_pos != -1) {
                col->pos[cur_block_id] = last_pos;
                col->value[cur_block_id] = last_value;
                cur_block_id++;
            }
	    last_pos = current_pos;
            last_value = 0;
        }
        unsigned long long mask = ((unsigned long long) 1) << (src_data[i] % BLOCK_BITS);
        last_value |= mask;
	    if(i == (src_length-1))
	    {
		    col->pos[cur_block_id] = last_pos;
		    col->value[cur_block_id] = last_value;
	    }
    }
}

gpu_boundary_matrix::gpu_boundary_matrix(phat::boundary_matrix <phat::vector_vector> *src_matrix,
                                         indx chunks_num, ScatterAllocator::AllocatorHandle allocator) {
    auto cols_num = (size_t) src_matrix->get_num_cols();
    auto h_matrix = new column[cols_num];
    auto h_chunk_offset = new indx[cols_num];
    auto h_column_type = new short[cols_num];
    auto h_column_length = new size_t[cols_num];
    auto h_dims = new dimension[cols_num];
    auto h_lowest_one_lookup = new indx[cols_num];
    auto h_chunks_start_offset = new indx[chunks_num + 1];
    auto h_is_reduced = new bool[cols_num];

    auto chunk_size = (size_t) CUDA_THREADS_EACH_BLOCK(cols_num);

    for (phat::index i = 0, chunk_pos = 0; i < cols_num; i++) {
        phat::column col;
        src_matrix->get_col(i, col);
        h_column_length[i] = col.size();
        h_column_type[i] = GLOBAL;
        h_lowest_one_lookup[i] = -1;
        h_is_reduced[i] = false;
        h_dims[i] = src_matrix->get_dim(i);
        if (i % chunk_size == 0) {
            h_chunks_start_offset[chunk_pos] = i;
            chunk_pos++;
        }
    }

    h_chunk_offset[chunks_num] = (indx)(cols_num);

    gpuErrchk(hipMalloc((void **) &matrix, sizeof(column) * cols_num));
    gpuErrchk(hipMalloc((void **) &chunk_offset, sizeof(indx) * (chunks_num + 1)));
    gpuErrchk(hipMalloc((void **) &chunk_columns_finished, sizeof(unsigned long long) * chunks_num));
    gpuErrchk(hipMalloc((void **) &column_type, sizeof(short) * cols_num));
    gpuErrchk(hipMalloc((void **) &dims, sizeof(dimension) * cols_num));
    gpuErrchk(hipMalloc((void **) &column_length, sizeof(size_t) * cols_num));
    gpuErrchk(hipMalloc((void **) &lowest_one_lookup, sizeof(indx) * cols_num));
    gpuErrchk(hipMalloc((void **) &is_active, sizeof(bool) * cols_num));
    gpuErrchk(hipMalloc((void **) &is_ready_for_mark, sizeof(bool) * cols_num));
    gpuErrchk(hipMalloc((void **) &is_reduced, sizeof(bool) * cols_num));

    size_t * d_column_length;
    gpuErrchk(hipMalloc((void **) &d_column_length, sizeof(size_t) * cols_num));
    gpuErrchk(hipMemcpy(d_column_length, h_column_length, sizeof(size_t) * cols_num, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(dims, h_dims, sizeof(dimension) * cols_num, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(column_length, h_column_length, sizeof(size_t) * cols_num, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(lowest_one_lookup, h_lowest_one_lookup, sizeof(indx) * cols_num, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(chunk_offset, h_chunk_offset, sizeof(indx) * (chunks_num + 1), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(column_type, h_column_type, sizeof(short) * cols_num, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(is_reduced, h_is_reduced, sizeof(bool) * cols_num, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(matrix, h_matrix, sizeof(column) * cols_num, hipMemcpyHostToDevice));

    indx ** tmp_gpu_columns, ** h_tmp_gpu_columns;
    h_tmp_gpu_columns = new indx * [cols_num];
    gpuErrchk(hipMalloc((void **) &tmp_gpu_columns, sizeof(indx *) * cols_num));
    allocate_all_columns <<< CUDA_BLOCKS_NUM(cols_num), CUDA_THREADS_EACH_BLOCK(cols_num) >>> (tmp_gpu_columns,
            d_column_length, cols_num, allocator);
    hipMemcpy(h_tmp_gpu_columns, tmp_gpu_columns, sizeof(indx *) * cols_num, hipMemcpyDeviceToHost);

    for (phat::index i = 0; i < cols_num; i++) {
        phat::column col;
        src_matrix->get_col(i, col);
        auto col_data_ptr = &col[0];

        gpuErrchk(hipMemcpy(h_tmp_gpu_columns[i], col_data_ptr, sizeof(indx) * col.size(),
                             hipMemcpyHostToDevice));
    }

    transform_all_columns <<< CUDA_BLOCKS_NUM(cols_num), CUDA_THREADS_EACH_BLOCK(cols_num) >>> (tmp_gpu_columns,
            column_length, matrix, cols_num, is_reduced, lowest_one_lookup, allocator);

    gpuErrchk(hipFree(tmp_gpu_columns));
    delete[] h_matrix;
    delete[] h_chunk_offset;
    delete[] h_column_type;
    delete[] h_column_length;
    delete[] h_dims;
    delete[] h_lowest_one_lookup;
    delete[] h_chunks_start_offset;
    delete[] h_tmp_gpu_columns;
}

__global__ void transform_unpacked_data(column *matrix, unpacked_matrix* u_matrix, ScatterAllocator::AllocatorHandle allocator)
{
    int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    //int block_id = blockDim.x;

    int count_length = 0;
    for(int i = 0; i<matrix[thread_id].data_length; i++)
    {
       // unsigned long long temp_value = matrix[thread_id].value[i];
        for(int j = 0; j < BLOCK_BITS; j++)
        {
            unsigned long long temp_value = 1 << j;
            if(matrix[thread_id].value[i] & temp_value)
            {
                count_length++;
            }
        }
    }

    u_matrix->column[thread_id].data = (indx*) allocator.malloc(sizeof(indx) * count_length);
    u_matrix->column[thread_id].data_length = count_length;
    count_length = 0;
    for(int i = 0; i < matrix[thread_id].data_length; i++)
    {
        for(int j = 0; j < BLOCK_BITS; j++)
        {
            unsigned long long temp_value = 1 << j;
            if(matrix[thread_id].value[i] & temp_value)
            {
                u_matrix->column[thread_id].data[count_length] = matrix[thread_id].pos[i] * BLOCK_BITS + j;
                count_length++;
            }
        }
    }
}

/*__host__ void free_cuda_memory(columns *matrix, dimension* dims, indx* lowest_one_lookup, indx* chunks_start_offset, indx* chunk_columns_finished, short* column_type, bool* is_active, bool* is_ready_for_mark) {
    gpuErrchk(hipFree(dims));
    gpuErrchk(hipFree(matrix));
    gpuErrchk(hipFree(lowest_one_lookup));
    gpuErrchk(hipFree(chunks_start_offset));
    gpuErrchk(hipFree(chunk_columns_finished));
    gpuErrchk(hipFree(column_type));
    gpuErrchk(hipFree(is_active));
    gpuErrchk(hipFree(is_ready_for_mark));
}
*/
__host__ void transfor_data_backto_cpu(phat::boundary_matrix<phat::vector_vector> *src_matrix,unpacked_matrix *d_matrix)
{
    indx cols_num = src_matrix->get_num_cols();
    auto h_all_columns = new unpacked_column[cols_num];

    gpuErrchk(hipMemcpy(h_all_columns, d_matrix->column, sizeof(column) * cols_num, hipMemcpyDeviceToHost));
    for (int i = 0; i < cols_num; i++) {
    auto h_single_column = h_all_columns[i];

    phat::column tmp_vector(h_single_column.data_length);
    gpuErrchk(hipMemcpy(&tmp_vector[0], h_single_column.data,
                sizeof(indx) * h_single_column.data_length, hipMemcpyDeviceToHost));
    src_matrix->set_col(i, tmp_vector);
    }

    delete [] h_all_columns;
}

__device__ dimension get_dim(dimension* dims, int col_id) {
    return dims[col_id];
}

__device__ bool is_empty(column* matrix, int col_id) {
    return matrix[col_id].data_length == 0 || matrix[col_id].value[0] == 0;
}

__device__ indx get_max_index(column* matrix, int col_id) {
    if (matrix[col_id].data_length == 0 || matrix[col_id].value[0] == 0)
        return -1;
    else {
        unsigned long long t = matrix[col_id].value[matrix[col_id].data_length - 1];
        int cnt = 0;
        while (t >> 1 != 0) {
            t = t >> 1;
            cnt++;
        }
        return (matrix[col_id].pos[matrix[col_id].data_length - 1] * BLOCK_BITS + cnt);
    }
}

__device__ void clear_column(column* matrix, int col_id) {
    matrix[col_id].data_length = 0;
}

__device__ void remove_max_index(column* matrix, int col) {
    if(matrix[col].data_length == 0)
        return;
    unsigned long long t = matrix[col].value[matrix[col].data_length - 1];
    int cnt = 1;
    while (t >> 1 != 0) {
        t = t >> 1;
        cnt++;
    }
    int tx = (1 << (cnt-1));
    matrix[col].value[matrix[col].data_length - 1]  ^= tx;
    if (matrix[col].value[matrix[col].data_length - 1] == 0)
        matrix[col].data_length--;
}

__device__ void check_lowest_one_locally(column* matrix, short* column_type, unsigned long long* chunk_columns_finished, dimension* dims,indx my_col_id, indx chunk_start, indx row_begin, indx num_cols, dimension cur_dim, indx *target_col, bool *ive_added) {
    if (cur_dim != get_dim(dims, my_col_id) || column_type[my_col_id] != GLOBAL) {
        if (!*ive_added) {
            atomicAdd((unsigned long long *) &chunk_columns_finished[0], (unsigned long long) 1);
            *ive_added = true;
        }
        return;
    }

    indx my_lowest_one = get_max_index(matrix, my_col_id);
    if (my_lowest_one >= row_begin) {
        for (indx col_id = chunk_start; col_id < num_cols; col_id++) {
            indx this_lowest_one = get_max_index(matrix, col_id);
            if (this_lowest_one == my_lowest_one) {
                *target_col = col_id;
                if (*ive_added) {
                    atomicAdd((unsigned long long *) &chunk_columns_finished[0], (unsigned long long) -1);
                }
                return;
            }
        }
        if (!*ive_added) {
            atomicAdd((unsigned long long *) &chunk_columns_finished[0], (unsigned long long) 1);
            *ive_added = true;
        }
    }
}

__device__ void add_two_columns(column* matrix, int target, int source, ScatterAllocator::AllocatorHandle allocator) {
    if(target == -1)
        return;
    size_t tgt_id = 0; size_t src_id = 0; size_t temp_id = 0;
    int msize = round_up_to_2s(matrix[target].data_length + matrix[source].data_length);
    auto new_pos = (indx *) allocator.malloc(sizeof(indx) * msize);
    auto new_value = (unsigned long long *) allocator.malloc(sizeof(unsigned long long) * msize);
    while (tgt_id < matrix[target].data_length && src_id < matrix[source].data_length) {
        if (matrix[target].pos[tgt_id] == matrix[source].pos[src_id]) {
            if (matrix[target].value[tgt_id] ^ matrix[source].value[src_id] != 0) {
                new_pos[temp_id] = matrix[target].pos[tgt_id];
                new_value[temp_id] = matrix[target].value[tgt_id] ^ matrix[source].value[src_id];
                temp_id++;
            }
            tgt_id++;
            src_id++;
        } else if (matrix[target].pos[tgt_id] < matrix[source].pos[src_id]) {
            if (matrix[target].pos[tgt_id] == matrix[source].pos[src_id + 1])
                tgt_id++;
            else {
                new_value[temp_id] = matrix[target].value[tgt_id];
                new_pos[temp_id] = matrix[target].pos[tgt_id];
                tgt_id++;
                temp_id++;
            }
        } else {
            if (matrix[target].pos[tgt_id + 1] == matrix[source].pos[src_id])
                src_id++;
            else {
                new_value[temp_id] = matrix[source].value[src_id];
                new_pos[temp_id] = matrix[source].pos[src_id];
                src_id++;
                temp_id++;
            }
        }
    }

    if (src_id < matrix[source].data_length) {
        memcpy(&new_value[temp_id], &matrix[source].value[src_id],
               sizeof(indx) * (matrix[source].data_length - src_id));
        memcpy(&new_pos[temp_id], &matrix[source].pos[temp_id],
               sizeof(indx) * (matrix[source].data_length - src_id));
        temp_id += matrix[source].data_length - src_id;
    } else if (tgt_id < matrix[target].data_length) {
        memcpy(&new_value[temp_id], &matrix[target].value[tgt_id],
               sizeof(indx) * (matrix[target].data_length - tgt_id));
        memcpy(&new_pos[temp_id], &matrix[target].pos[tgt_id],
               sizeof(indx) * (matrix[target].data_length - tgt_id));
        temp_id += matrix[target].data_length - tgt_id;
    }
    matrix[target].pos = new_pos;
    matrix[target].value = new_value;
    matrix[target].data_length = temp_id;
}

__device__ void mark_and_clean(column* matrix, indx* lowest_one_lookup, short* column_type, dimension* dims, indx my_col_id, indx row_begin, dimension cur_dim) {
    if (cur_dim != get_dim(dims, my_col_id) || column_type[my_col_id] != GLOBAL) {
        return;
    }

    indx my_lowest_one = get_max_index(matrix, my_col_id);
    if (lowest_one_lookup[my_lowest_one] == -1 && my_lowest_one >= row_begin) {
      	lowest_one_lookup[my_lowest_one] = my_col_id;
        column_type[my_col_id] = LOCAL_NEGATIVE;
        column_type[my_lowest_one] = LOCAL_POSITIVE;
        clear_column(matrix, my_lowest_one);
    }
}

